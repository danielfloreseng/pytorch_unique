
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

#include <thrust/unique.h>

THCTensor *unique_kernel(THCState *state, THCudaLongTensor *index, THCTensor *input) {
  #if CUDA_VERSION >= 7000
    THCThrustAllocator thrustAlloc(state);
  #define THRUST_EXEC(fn, ...) fn(thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)), ##__VA_ARGS__)
  #else
  #define THRUST_EXEC(fn, ...) fn(##__VA_ARGS__)
  #endif

  return NULL;
}

#endif
