
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/sort.h>

THCTensor *unique_kernel(THCState *state, THCudaLongTensor *index, THCTensor *input) {
  /* index = THCudaLongTensor_newContiguous(state, index); */
  input = THCTensor_(newContiguous)(state, input);
  THCTensor *output = input;

  thrust::device_ptr<real> output_data(THCTensor_(data)(state, input));
  ptrdiff_t size = THCTensor_(nElement)(state, output);

  THRUST_ALLOC(state);
  THRUST_EXEC(thrust::sort, output_data, output_data + size);

  THCTensor_(free)(state, input);

  return NULL;
  /* return output; */
}

#endif
