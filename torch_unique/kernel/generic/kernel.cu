
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

void unique_(single)(THCState *state, THCTensor *input) {
  input = THCTensor_(newContiguous)(state, input);

  thrust::device_ptr<real> first(THCTensor_(data)(state, input));
  ptrdiff_t numel = THCTensor_(nElement)(state, input);

  THRUST_ALLOC(state);
  THRUST_EXEC(thrust::sort, first, first + numel);
  thrust::device_ptr<real> last(THRUST_EXEC(thrust::unique, first, first + numel));

  numel = last - first;
  THCTensor_(resize1d)(state, input, numel);

  THCTensor_(free)(state, input);
}

void unique_(byKey)(THCState *state, THCTensor *key, THCTensor *value) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, key, value));

  key = THCTensor_(newContiguous)(state, key);
  value = THCTensor_(newContiguous)(state, value);

  thrust::device_ptr<real> firstKey(THCTensor_(data)(state, key));
  thrust::device_ptr<real> firstValue(THCTensor_(data)(state, value));
  ptrdiff_t numel = THCTensor_(nElement)(state, key);

  THRUST_ALLOC(state);
  THRUST_EXEC(thrust::sort_by_key, firstKey, firstKey + numel, firstValue);

  thrust::pair<thrust::device_ptr<real>, thrust::device_ptr<real> > last(THRUST_EXEC(thrust::unique_by_key, firstKey, firstKey + numel, firstValue));

  THCTensor_(resize1d)(state, key, last.first - firstKey);
  THCTensor_(resize1d)(state, value, last.second - firstValue);

  THCTensor_(free)(state, key);
  THCTensor_(free)(state, value);
}

#endif
