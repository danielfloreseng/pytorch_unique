
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

THCTensor *unique_kernel(THCState *state, THCudaLongTensor *index, THCTensor *input) {
  input = THCTensor_(newContiguous)(state, input);

  thrust::device_ptr<real> idxThrust(THCTensor_(data)(state, input));
  ptrdiff_t numel = THCTensor_(nElement)(state, input);
  THRUST_ALLOC(state);
  THRUST_EXEC(thrust::sort, idxThrust, idxThrust + numel);
  thrust::device_ptr<real> endIdxThrust(THRUST_EXEC(thrust::unique, idxThrust, idxThrust + numel));
  numel = endIdxThrust - idxThrust;
  THCTensor_(resize1d)(state, input, numel);

  THCTensor_(free)(state, input);

  return input;
}

#endif
