
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

void unique_(single)(THCState *state, THCTensor *input) {
  input = THCTensor_(newContiguous)(state, input);

  thrust::device_ptr<real> first(THCTensor_(data)(state, input));
  ptrdiff_t numel = THCTensor_(nElement)(state, input);
  THRUST_ALLOC(state);
  THRUST_EXEC(thrust::sort, first, first + numel);
  thrust::device_ptr<real> last(THRUST_EXEC(thrust::unique, first, first + numel));
  numel = last - first;
  THCTensor_(resize1d)(state, input, numel);

  THCTensor_(free)(state, input);
}

void unique_(byKey)(single)(THCState *state, THCTensor *key, THCTensor *value) {
  key = THCTensor_(newContiguous)(state, key);
  value = THCTensor_(newContiguous)(state, value);

  thrust::device_ptr<real> firstKey(THCTensor_(data)(state, key));
  thrust::device_ptr<real> firstValue(THCTensor_(data)(state, value));
  ptrdiff_t numel = THCTensor_(nElement)(state, key);
  THRUST_ALLOC(state);
  THRUST_EXEC(thrust::sort_by_key, firstKey, firstKey + numel, firstValue);
  thrust::device_ptr<real> last(THRUST_EXEC(thrust::unique_by_key, firstKey, firstKey + numel, firstValue));
  numel = last - first;
  THCTensor_(resize1d)(state, key, numel);
  THCTensor_(resize1d)(state, value, numel);

  THCTensor_(free)(state, key);
  THCTensor_(free)(state, value);
}

#endif
